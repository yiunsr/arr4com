
#include <hip/hip_runtime.h>

extern "C" __global__ void a4c_addf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] + y[i];
    }
}

extern "C" __global__ void a4c_subf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] - y[i];
    }
}

extern "C" __global__ void a4c_mulf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] * y[i];
    }
}

extern "C" __global__ void a4c_divf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] / y[i];
    }
}

extern "C" __global__ void a4c_cosf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = cosf(x[i]);
    }
}

extern "C" __global__ void a4c_sinf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = sinf(x[i]);
    }
}

extern "C" __global__ void a4c_tanf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = tanf(x[i]);
    }
}

extern "C" __global__ void a4c_acosf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = acosf(x[i]);
    }
}

extern "C" __global__ void a4c_asinf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = asinf(x[i]);
    }
}

extern "C" __global__ void a4c_atanf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = atanf(x[i]);
    }
}

extern "C" __global__ void a4c_coshf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = coshf(x[i]);
    }
}

extern "C" __global__ void a4c_sinhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = sinhf(x[i]);
    }
}

extern "C" __global__ void a4c_tanhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = tanhf(x[i]);
    }
}

extern "C" __global__ void a4c_acoshf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = acoshf(x[i]);
    }
}

extern "C" __global__ void a4c_asinhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = asinhf(x[i]);
    }
}

extern "C" __global__ void a4c_atanhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = atanhf(x[i]);
    }
}

extern "C" __global__ void a4c_lnf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = logf(x[i]);
    }
}
extern "C" __global__ void a4c_ln_1pf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = log1pf(x[i]);
    }
}
extern "C" __global__ void a4c_log10f32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = log10f(x[i]);
    }
}
extern "C" __global__ void a4c_log2f32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = log2f(x[i]);
    }
}

