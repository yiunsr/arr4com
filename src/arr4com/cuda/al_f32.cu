
#include <hip/hip_runtime.h>

extern "C" __global__ void a4c_addf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] + y[i];
    }
}

extern "C" __global__ void a4c_subf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] - y[i];
    }
}

extern "C" __global__ void a4c_mulf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] * y[i];
    }
}

extern "C" __global__ void a4c_divf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] / y[i];
    }
}

extern "C" __global__ void a4c_sinf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = sinf(x[i]);
    }
}

extern "C" __global__ void a4c_cosf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = cosf(x[i]);
    }
}

extern "C" __global__ void a4c_tanf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = tanf(x[i]);
    }
}
