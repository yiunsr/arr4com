#include "hip/hip_runtime.h"

extern "C" __global__ void arr4com_sort(const float* x, const float* y, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] + y[i];
    }
}
