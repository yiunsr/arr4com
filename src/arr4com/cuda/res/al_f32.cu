
#include <hip/hip_runtime.h>

extern "C" __global__ void a4c_addf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] + y[i];
    }
}

extern "C" __global__ void a4c_subf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] - y[i];
    }
}

extern "C" __global__ void a4c_mulf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] * y[i];
    }
}

extern "C" __global__ void a4c_divf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] / y[i];
    }
}

extern "C" __global__ void a4c_mul_addf32(const float* x, const float* y, const float* z, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = fmaf(x[i], y[i], z[i]);
    }
}

extern "C" __global__ void a4c_gtff32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] > y[i];
    }
}
extern "C" __global__ void a4c_gteff32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] >= y[i];
    }
}
extern "C" __global__ void a4c_ltff32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] < y[i];
    }
}
extern "C" __global__ void a4c_lteff32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] <= y[i];
    }
}

extern "C" __global__ void a4c_ceilf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = ceilf(x[i]);
    }
}
extern "C" __global__ void a4c_floorf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = floorf(x[i]);
    }
}
extern "C" __global__ void a4c_roundf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = roundf (x[i]);
    }
}
extern "C" __global__ void a4c_truncf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = truncf(x[i]);
    }
}
extern "C" __global__ void a4c_absf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = fabsf(x[i]);
    }
}
extern "C" __global__ void a4c_maxf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = fmaxf(x[i], y[i]);
    }
}
extern "C" __global__ void a4c_minf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = fminf(x[i], y[i]);
    }
}
extern "C" __global__ void a4c_copysignf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = copysignf(x[i], y[i]);
    }
}

extern "C" __global__ void a4c_cosf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = cosf(x[i]);
    }
}

extern "C" __global__ void a4c_sinf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = sinf(x[i]);
    }
}

extern "C" __global__ void a4c_tanf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = tanf(x[i]);
    }
}

extern "C" __global__ void a4c_acosf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = acosf(x[i]);
    }
}

extern "C" __global__ void a4c_asinf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = asinf(x[i]);
    }
}

extern "C" __global__ void a4c_atanf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = atanf(x[i]);
    }
}

extern "C" __global__ void a4c_atan2f32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = atan2f(x[i], y[i]);
    }
}

extern "C" __global__ void a4c_coshf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = coshf(x[i]);
    }
}

extern "C" __global__ void a4c_sinhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = sinhf(x[i]);
    }
}

extern "C" __global__ void a4c_tanhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = tanhf(x[i]);
    }
}

extern "C" __global__ void a4c_acoshf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = acoshf(x[i]);
    }
}

extern "C" __global__ void a4c_asinhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = asinhf(x[i]);
    }
}

extern "C" __global__ void a4c_atanhf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = atanhf(x[i]);
    }
}

extern "C" __global__ void a4c_lnf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = logf(x[i]);
    }
}
extern "C" __global__ void a4c_ln_1pf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = log1pf(x[i]);
    }
}
extern "C" __global__ void a4c_log10f32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = log10f(x[i]);
    }
}
extern "C" __global__ void a4c_log2f32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = log2f(x[i]);
    }
}

extern "C" __global__ void a4c_expf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = expf (x[i]);
    }
}
extern "C" __global__ void a4c_exp2f32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = exp2f(x[i]);
    }
}
extern "C" __global__ void a4c_exp_m1f32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = expm1f(x[i]);
    }
}

extern "C" __global__ void a4c_sqrtf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = sqrtf (x[i]);
    }
}
extern "C" __global__ void a4c_cbrtf32(const float* x, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = cbrtf (x[i]);
    }
}

extern "C" __global__ void a4c_powff32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = powf(x[i], y[i]);
    }
}
extern "C" __global__ void a4c_hypotf32(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = hypotf(x[i], y[i]);
    }
}